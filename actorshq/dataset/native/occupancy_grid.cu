#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>
#include <ATen/ATen.h>
#include <vector>
#include <utils.cuh>

class OccupanyGrid
{
    public:
        OccupanyGrid(const unsigned long grid_resolution, const int buffer_size)
            : m_grid_resolution(grid_resolution)
            , m_grid_extent({grid_resolution, grid_resolution, grid_resolution})
            , m_buffer_size(buffer_size)
            , m_available_buffer_index(0)
        {
            for (int i = 0; i < m_buffer_size; ++i)
            {
                hipChannelFormatDesc channel_desc = hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindUnsigned);
                hipArray_t grid_array = nullptr;
                hipMalloc3DArray(&grid_array, &channel_desc, m_grid_extent);

                hipResourceDesc resource_desc;
                memset(&resource_desc, 0, sizeof(resource_desc));
                resource_desc.resType = hipResourceTypeArray;
                resource_desc.res.array.array = grid_array;

                hipTextureDesc texture_desc;
                memset(&texture_desc, 0, sizeof(texture_desc));
                texture_desc.addressMode[0] = hipAddressModeClamp;
                texture_desc.addressMode[1] = hipAddressModeClamp;
                texture_desc.addressMode[2] = hipAddressModeClamp;
                texture_desc.filterMode = hipFilterModeLinear;
                texture_desc.readMode = hipReadModeNormalizedFloat;
                texture_desc.normalizedCoords = 1;

                hipTextureObject_t texture_object = 0;
                hipCreateTextureObject(&texture_object, &resource_desc, &texture_desc, nullptr);

                m_grid_arrays.push_back(grid_array);
                m_texture_objects.push_back(texture_object);
            }
        }

        ~OccupanyGrid()
        {
            for (const auto& m_grid_array : m_grid_arrays)
            {
                hipFreeArray(m_grid_array);
            }
            for (const auto& m_texture_object : m_texture_objects)
            {
                hipDestroyTextureObject(m_texture_object);
            }
        }

        hipTextureObject_t add_grid(torch::Tensor grid)
        {
            CHECK_CONTIGUITY_AND_DEVICE(grid, torch::kCUDA);
            if (grid.size(0) != m_grid_resolution || grid.size(1) != m_grid_resolution || grid.size(2) != m_grid_resolution)
            {
                throw std::runtime_error("Provided grid doesn't have the correct resolution!");
            }

            auto used_buffer_index = m_available_buffer_index;
            m_available_buffer_index = (m_available_buffer_index + 1) % m_buffer_size;

            hipMemcpy3DParms memcpy_params = {0};
            memcpy_params.srcPtr.pitch = m_grid_resolution * sizeof(uint8_t);
            memcpy_params.srcPtr.xsize = m_grid_resolution;
            memcpy_params.srcPtr.ysize = m_grid_resolution;
            memcpy_params.srcPtr.ptr = grid.data_ptr<uint8_t>();
            memcpy_params.dstArray = m_grid_arrays[used_buffer_index];
            memcpy_params.extent = m_grid_extent;
            memcpy_params.kind = hipMemcpyDefault;
            hipMemcpy3D(&memcpy_params);

            return m_texture_objects[used_buffer_index];
        }

    private:
        const unsigned long m_grid_resolution;
        const int m_buffer_size;
        int m_available_buffer_index;
        hipExtent m_grid_extent;
        std::vector<hipTextureObject_t> m_texture_objects;
        std::vector<hipArray_t> m_grid_arrays;
};

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m)
{
    py::class_<OccupanyGrid>(m, "OccupanyGrid")
        .def(py::init<const unsigned long, const int>())
        .def("add_grid", &OccupanyGrid::add_grid);
}
